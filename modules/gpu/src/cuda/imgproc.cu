#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/vec_traits.hpp"
#include "opencv2/core/cuda/vec_math.hpp"
#include "opencv2/core/cuda/saturate_cast.hpp"
#include "opencv2/core/cuda/border_interpolate.hpp"
#include "internal_shared.hpp"

namespace cv { namespace gpu { namespace cudev
{
    namespace imgproc
    {
        /////////////////////////////////// MeanShiftfiltering ///////////////////////////////////////////////

        texture<uchar4, 2> tex_meanshift;

        __device__ short2 do_mean_shift(int x0, int y0, unsigned char* out,
                                        size_t out_step, int cols, int rows,
                                        int sp, int sr, int maxIter, float eps)
        {
            int isr2 = sr*sr;
            uchar4 c = tex2D(tex_meanshift, x0, y0 );

            // iterate meanshift procedure
            for( int iter = 0; iter < maxIter; iter++ )
            {
                int count = 0;
                int s0 = 0, s1 = 0, s2 = 0, sx = 0, sy = 0;
                float icount;

                //mean shift: process pixels in window (p-sigmaSp)x(p+sigmaSp)
                int minx = x0-sp;
                int miny = y0-sp;
                int maxx = x0+sp;
                int maxy = y0+sp;

                for( int y = miny; y <= maxy; y++)
                {
                    int rowCount = 0;
                    for( int x = minx; x <= maxx; x++ )
                    {
                        uchar4 t = tex2D( tex_meanshift, x, y );

                        int norm2 = (t.x - c.x) * (t.x - c.x) + (t.y - c.y) * (t.y - c.y) + (t.z - c.z) * (t.z - c.z);
                        if( norm2 <= isr2 )
                        {
                            s0 += t.x; s1 += t.y; s2 += t.z;
                            sx += x; rowCount++;
                        }
                    }
                    count += rowCount;
                    sy += y*rowCount;
                }

                if( count == 0 )
                    break;

                icount = 1.f/count;
                int x1 = __float2int_rz(sx*icount);
                int y1 = __float2int_rz(sy*icount);
                s0 = __float2int_rz(s0*icount);
                s1 = __float2int_rz(s1*icount);
                s2 = __float2int_rz(s2*icount);

                int norm2 = (s0 - c.x) * (s0 - c.x) + (s1 - c.y) * (s1 - c.y) + (s2 - c.z) * (s2 - c.z);

                bool stopFlag = (x0 == x1 && y0 == y1) || (::abs(x1-x0) + ::abs(y1-y0) + norm2 <= eps);

                x0 = x1; y0 = y1;
                c.x = s0; c.y = s1; c.z = s2;

                if( stopFlag )
                    break;
            }

            int base = (blockIdx.y * blockDim.y + threadIdx.y) * out_step + (blockIdx.x * blockDim.x + threadIdx.x) * 4 * sizeof(uchar);
            *(uchar4*)(out + base) = c;

            return make_short2((short)x0, (short)y0);
        }

        __global__ void meanshift_kernel(unsigned char* out, size_t out_step, int cols, int rows, int sp, int sr, int maxIter, float eps )
        {
            int x0 = blockIdx.x * blockDim.x + threadIdx.x;
            int y0 = blockIdx.y * blockDim.y + threadIdx.y;

            if( x0 < cols && y0 < rows )
                do_mean_shift(x0, y0, out, out_step, cols, rows, sp, sr, maxIter, eps);
        }

        __global__ void meanshiftproc_kernel(unsigned char* outr, size_t outrstep,
                                             unsigned char* outsp, size_t outspstep,
                                             int cols, int rows,
                                             int sp, int sr, int maxIter, float eps)
        {
            int x0 = blockIdx.x * blockDim.x + threadIdx.x;
            int y0 = blockIdx.y * blockDim.y + threadIdx.y;

            if( x0 < cols && y0 < rows )
            {
                int basesp = (blockIdx.y * blockDim.y + threadIdx.y) * outspstep + (blockIdx.x * blockDim.x + threadIdx.x) * 2 * sizeof(short);
                *(short2*)(outsp + basesp) = do_mean_shift(x0, y0, outr, outrstep, cols, rows, sp, sr, maxIter, eps);
            }
        }

        void meanShiftFiltering_gpu(const PtrStepSzb& src, PtrStepSzb dst, int sp, int sr, int maxIter, float eps, hipStream_t stream)
        {
            dim3 grid(1, 1, 1);
            dim3 threads(32, 8, 1);
            grid.x = divUp(src.cols, threads.x);
            grid.y = divUp(src.rows, threads.y);

            hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>();
            cudaSafeCall( hipBindTexture2D( 0, tex_meanshift, src.data, desc, src.cols, src.rows, src.step ) );

            meanshift_kernel<<< grid, threads, 0, stream >>>( dst.data, dst.step, dst.cols, dst.rows, sp, sr, maxIter, eps );
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );

            //cudaSafeCall( hipUnbindTexture( tex_meanshift ) );
        }

        void meanShiftProc_gpu(const PtrStepSzb& src, PtrStepSzb dstr, PtrStepSzb dstsp, int sp, int sr, int maxIter, float eps, hipStream_t stream)
        {
            dim3 grid(1, 1, 1);
            dim3 threads(32, 8, 1);
            grid.x = divUp(src.cols, threads.x);
            grid.y = divUp(src.rows, threads.y);

            hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>();
            cudaSafeCall( hipBindTexture2D( 0, tex_meanshift, src.data, desc, src.cols, src.rows, src.step ) );

            meanshiftproc_kernel<<< grid, threads, 0, stream >>>( dstr.data, dstr.step, dstsp.data, dstsp.step, dstr.cols, dstr.rows, sp, sr, maxIter, eps );
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );

            //cudaSafeCall( hipUnbindTexture( tex_meanshift ) );
        }

        /////////////////////////////////// drawColorDisp ///////////////////////////////////////////////

        template <typename T>
        __device__ unsigned int cvtPixel(T d, int ndisp, float S = 1, float V = 1)
        {
            unsigned int H = ((ndisp-d) * 240)/ndisp;

            unsigned int hi = (H/60) % 6;
            float f = H/60.f - H/60;
            float p = V * (1 - S);
            float q = V * (1 - f * S);
            float t = V * (1 - (1 - f) * S);

            float3 res;

            if (hi == 0) //R = V,	G = t,	B = p
            {
                res.x = p;
                res.y = t;
                res.z = V;
            }

            if (hi == 1) // R = q,	G = V,	B = p
            {
                res.x = p;
                res.y = V;
                res.z = q;
            }

            if (hi == 2) // R = p,	G = V,	B = t
            {
                res.x = t;
                res.y = V;
                res.z = p;
            }

            if (hi == 3) // R = p,	G = q,	B = V
            {
                res.x = V;
                res.y = q;
                res.z = p;
            }

            if (hi == 4) // R = t,	G = p,	B = V
            {
                res.x = V;
                res.y = p;
                res.z = t;
            }

            if (hi == 5) // R = V,	G = p,	B = q
            {
                res.x = q;
                res.y = p;
                res.z = V;
            }
            const unsigned int b = (unsigned int)(::max(0.f, ::min(res.x, 1.f)) * 255.f);
            const unsigned int g = (unsigned int)(::max(0.f, ::min(res.y, 1.f)) * 255.f);
            const unsigned int r = (unsigned int)(::max(0.f, ::min(res.z, 1.f)) * 255.f);
            const unsigned int a = 255U;

            return (a << 24) + (r << 16) + (g << 8) + b;
        }

        __global__ void drawColorDisp(uchar* disp, size_t disp_step, uchar* out_image, size_t out_step, int width, int height, int ndisp)
        {
            const int x = (blockIdx.x * blockDim.x + threadIdx.x) << 2;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if(x < width && y < height)
            {
                uchar4 d4 = *(uchar4*)(disp + y * disp_step + x);

                uint4 res;
                res.x = cvtPixel(d4.x, ndisp);
                res.y = cvtPixel(d4.y, ndisp);
                res.z = cvtPixel(d4.z, ndisp);
                res.w = cvtPixel(d4.w, ndisp);

                uint4* line = (uint4*)(out_image + y * out_step);
                line[x >> 2] = res;
            }
        }

        __global__ void drawColorDisp(short* disp, size_t disp_step, uchar* out_image, size_t out_step, int width, int height, int ndisp)
        {
            const int x = (blockIdx.x * blockDim.x + threadIdx.x) << 1;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if(x < width && y < height)
            {
                short2 d2 = *(short2*)(disp + y * disp_step + x);

                uint2 res;
                res.x = cvtPixel(d2.x, ndisp);
                res.y = cvtPixel(d2.y, ndisp);

                uint2* line = (uint2*)(out_image + y * out_step);
                line[x >> 1] = res;
            }
        }


        void drawColorDisp_gpu(const PtrStepSzb& src, const PtrStepSzb& dst, int ndisp, const hipStream_t& stream)
        {
            dim3 threads(16, 16, 1);
            dim3 grid(1, 1, 1);
            grid.x = divUp(src.cols, threads.x << 2);
            grid.y = divUp(src.rows, threads.y);

            drawColorDisp<<<grid, threads, 0, stream>>>(src.data, src.step, dst.data, dst.step, src.cols, src.rows, ndisp);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        void drawColorDisp_gpu(const PtrStepSz<short>& src, const PtrStepSzb& dst, int ndisp, const hipStream_t& stream)
        {
            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);
            grid.x = divUp(src.cols, threads.x << 1);
            grid.y = divUp(src.rows, threads.y);

            drawColorDisp<<<grid, threads, 0, stream>>>(src.data, src.step / sizeof(short), dst.data, dst.step, src.cols, src.rows, ndisp);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        /////////////////////////////////// reprojectImageTo3D ///////////////////////////////////////////////

        __constant__ float cq[16];

        template <typename T, typename D>
        __global__ void reprojectImageTo3D(const PtrStepSz<T> disp, PtrStep<D> xyz)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (y >= disp.rows || x >= disp.cols)
                return;

            const float qx = x * cq[ 0] + y * cq[ 1] + cq[ 3];
            const float qy = x * cq[ 4] + y * cq[ 5] + cq[ 7];
            const float qz = x * cq[ 8] + y * cq[ 9] + cq[11];
            const float qw = x * cq[12] + y * cq[13] + cq[15];

            const T d = disp(y, x);

            const float iW = 1.f / (qw + cq[14] * d);

            D v = VecTraits<D>::all(1.0f);
            v.x = (qx + cq[2] * d) * iW;
            v.y = (qy + cq[6] * d) * iW;
            v.z = (qz + cq[10] * d) * iW;

            xyz(y, x) = v;
        }

        template <typename T, typename D>
        void reprojectImageTo3D_gpu(const PtrStepSzb disp, PtrStepSzb xyz, const float* q, hipStream_t stream)
        {
            dim3 block(32, 8);
            dim3 grid(divUp(disp.cols, block.x), divUp(disp.rows, block.y));

            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cq), q, 16 * sizeof(float)) );

            reprojectImageTo3D<T, D><<<grid, block, 0, stream>>>((PtrStepSz<T>)disp, (PtrStepSz<D>)xyz);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template void reprojectImageTo3D_gpu<uchar, float3>(const PtrStepSzb disp, PtrStepSzb xyz, const float* q, hipStream_t stream);
        template void reprojectImageTo3D_gpu<uchar, float4>(const PtrStepSzb disp, PtrStepSzb xyz, const float* q, hipStream_t stream);
        template void reprojectImageTo3D_gpu<short, float3>(const PtrStepSzb disp, PtrStepSzb xyz, const float* q, hipStream_t stream);
        template void reprojectImageTo3D_gpu<short, float4>(const PtrStepSzb disp, PtrStepSzb xyz, const float* q, hipStream_t stream);

        /////////////////////////////////////////// Corner Harris /////////////////////////////////////////////////

        texture<float, hipTextureType2D, hipReadModeElementType> harrisDxTex(0, hipFilterModePoint, hipAddressModeClamp);
        texture<float, hipTextureType2D, hipReadModeElementType> harrisDyTex(0, hipFilterModePoint, hipAddressModeClamp);

        __global__ void cornerHarris_kernel(const int block_size, const float k, PtrStepSzf dst)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x < dst.cols && y < dst.rows)
            {
                float a = 0.f;
                float b = 0.f;
                float c = 0.f;

                const int ibegin = y - (block_size / 2);
                const int jbegin = x - (block_size / 2);
                const int iend = ibegin + block_size;
                const int jend = jbegin + block_size;

                for (int i = ibegin; i < iend; ++i)
                {
                    for (int j = jbegin; j < jend; ++j)
                    {
                        float dx = tex2D(harrisDxTex, j, i);
                        float dy = tex2D(harrisDyTex, j, i);

                        a += dx * dx;
                        b += dx * dy;
                        c += dy * dy;
                    }
                }

                dst(y, x) = a * c - b * b - k * (a + c) * (a + c);
            }
        }

        template <typename BR, typename BC>
        __global__ void cornerHarris_kernel(const int block_size, const float k, PtrStepSzf dst, const BR border_row, const BC border_col)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x < dst.cols && y < dst.rows)
            {
                float a = 0.f;
                float b = 0.f;
                float c = 0.f;

                const int ibegin = y - (block_size / 2);
                const int jbegin = x - (block_size / 2);
                const int iend = ibegin + block_size;
                const int jend = jbegin + block_size;

                for (int i = ibegin; i < iend; ++i)
                {
                    const int y = border_col.idx_row(i);

                    for (int j = jbegin; j < jend; ++j)
                    {
                        const int x = border_row.idx_col(j);

                        float dx = tex2D(harrisDxTex, x, y);
                        float dy = tex2D(harrisDyTex, x, y);

                        a += dx * dx;
                        b += dx * dy;
                        c += dy * dy;
                    }
                }

                dst(y, x) = a * c - b * b - k * (a + c) * (a + c);
            }
        }

        void cornerHarris_gpu(int block_size, float k, PtrStepSzf Dx, PtrStepSzf Dy, PtrStepSzf dst, int border_type, hipStream_t stream)
        {
            dim3 block(32, 8);
            dim3 grid(divUp(Dx.cols, block.x), divUp(Dx.rows, block.y));

            bindTexture(&harrisDxTex, Dx);
            bindTexture(&harrisDyTex, Dy);

            switch (border_type)
            {
            case BORDER_REFLECT101_GPU:
                cornerHarris_kernel<<<grid, block, 0, stream>>>(block_size, k, dst, BrdRowReflect101<void>(Dx.cols), BrdColReflect101<void>(Dx.rows));
                break;

            case BORDER_REFLECT_GPU:
                cornerHarris_kernel<<<grid, block, 0, stream>>>(block_size, k, dst, BrdRowReflect<void>(Dx.cols), BrdColReflect<void>(Dx.rows));
                break;

            case BORDER_REPLICATE_GPU:
                cornerHarris_kernel<<<grid, block, 0, stream>>>(block_size, k, dst);
                break;
            }

            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        /////////////////////////////////////////// Corner Min Eigen Val /////////////////////////////////////////////////

        texture<float, hipTextureType2D, hipReadModeElementType> minEigenValDxTex(0, hipFilterModePoint, hipAddressModeClamp);
        texture<float, hipTextureType2D, hipReadModeElementType> minEigenValDyTex(0, hipFilterModePoint, hipAddressModeClamp);

        __global__ void cornerMinEigenVal_kernel(const int block_size, PtrStepSzf dst)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x < dst.cols && y < dst.rows)
            {
                float a = 0.f;
                float b = 0.f;
                float c = 0.f;

                const int ibegin = y - (block_size / 2);
                const int jbegin = x - (block_size / 2);
                const int iend = ibegin + block_size;
                const int jend = jbegin + block_size;

                for (int i = ibegin; i < iend; ++i)
                {
                    for (int j = jbegin; j < jend; ++j)
                    {
                        float dx = tex2D(minEigenValDxTex, j, i);
                        float dy = tex2D(minEigenValDyTex, j, i);

                        a += dx * dx;
                        b += dx * dy;
                        c += dy * dy;
                    }
                }

                a *= 0.5f;
                c *= 0.5f;

                dst(y, x) = (a + c) - sqrtf((a - c) * (a - c) + b * b);
            }
        }


        template <typename BR, typename BC>
        __global__ void cornerMinEigenVal_kernel(const int block_size, PtrStepSzf dst, const BR border_row, const BC border_col)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x < dst.cols && y < dst.rows)
            {
                float a = 0.f;
                float b = 0.f;
                float c = 0.f;

                const int ibegin = y - (block_size / 2);
                const int jbegin = x - (block_size / 2);
                const int iend = ibegin + block_size;
                const int jend = jbegin + block_size;

                for (int i = ibegin; i < iend; ++i)
                {
                    int y = border_col.idx_row(i);

                    for (int j = jbegin; j < jend; ++j)
                    {
                        int x = border_row.idx_col(j);

                        float dx = tex2D(minEigenValDxTex, x, y);
                        float dy = tex2D(minEigenValDyTex, x, y);

                        a += dx * dx;
                        b += dx * dy;
                        c += dy * dy;
                    }
                }

                a *= 0.5f;
                c *= 0.5f;

                dst(y, x) = (a + c) - sqrtf((a - c) * (a - c) + b * b);
            }
        }

        void cornerMinEigenVal_gpu(int block_size, PtrStepSzf Dx, PtrStepSzf Dy, PtrStepSzf dst, int border_type, hipStream_t stream)
        {
            dim3 block(32, 8);
            dim3 grid(divUp(Dx.cols, block.x), divUp(Dx.rows, block.y));

            bindTexture(&minEigenValDxTex, Dx);
            bindTexture(&minEigenValDyTex, Dy);

            switch (border_type)
            {
            case BORDER_REFLECT101_GPU:
                cornerMinEigenVal_kernel<<<grid, block, 0, stream>>>(block_size, dst, BrdRowReflect101<void>(Dx.cols), BrdColReflect101<void>(Dx.rows));
                break;

            case BORDER_REFLECT_GPU:
                cornerMinEigenVal_kernel<<<grid, block, 0, stream>>>(block_size, dst, BrdRowReflect<void>(Dx.cols), BrdColReflect<void>(Dx.rows));
                break;

            case BORDER_REPLICATE_GPU:
                cornerMinEigenVal_kernel<<<grid, block, 0, stream>>>(block_size, dst);
                break;
            }

            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall(hipDeviceSynchronize());
        }

        //////////////////////////////////////////////////////////////////////////
        // buildWarpMaps

        // TODO use intrinsics like __sinf and so on

        namespace build_warp_maps
        {

            __constant__ float ck_rinv[9];
            __constant__ float cr_kinv[9];
            __constant__ float ct[3];
            __constant__ float cscale;
        }


        class PlaneMapper
        {
        public:
            static __device__ __forceinline__ void mapBackward(float u, float v, float &x, float &y)
            {
                using namespace build_warp_maps;

                float x_ = u / cscale - ct[0];
                float y_ = v / cscale - ct[1];

                float z;
                x = ck_rinv[0] * x_ + ck_rinv[1] * y_ + ck_rinv[2] * (1 - ct[2]);
                y = ck_rinv[3] * x_ + ck_rinv[4] * y_ + ck_rinv[5] * (1 - ct[2]);
                z = ck_rinv[6] * x_ + ck_rinv[7] * y_ + ck_rinv[8] * (1 - ct[2]);

                x /= z;
                y /= z;
            }
        };


        class CylindricalMapper
        {
        public:
            static __device__ __forceinline__ void mapBackward(float u, float v, float &x, float &y)
            {
                using namespace build_warp_maps;

                u /= cscale;
                float x_ = ::sinf(u);
                float y_ = v / cscale;
                float z_ = ::cosf(u);

                float z;
                x = ck_rinv[0] * x_ + ck_rinv[1] * y_ + ck_rinv[2] * z_;
                y = ck_rinv[3] * x_ + ck_rinv[4] * y_ + ck_rinv[5] * z_;
                z = ck_rinv[6] * x_ + ck_rinv[7] * y_ + ck_rinv[8] * z_;

                if (z > 0) { x /= z; y /= z; }
                else x = y = -1;
            }
        };


        class SphericalMapper
        {
        public:
            static __device__ __forceinline__ void mapBackward(float u, float v, float &x, float &y)
            {
                using namespace build_warp_maps;

                v /= cscale;
                u /= cscale;

                float sinv = ::sinf(v);
                float x_ = sinv * ::sinf(u);
                float y_ = -::cosf(v);
                float z_ = sinv * ::cosf(u);

                float z;
                x = ck_rinv[0] * x_ + ck_rinv[1] * y_ + ck_rinv[2] * z_;
                y = ck_rinv[3] * x_ + ck_rinv[4] * y_ + ck_rinv[5] * z_;
                z = ck_rinv[6] * x_ + ck_rinv[7] * y_ + ck_rinv[8] * z_;

                if (z > 0) { x /= z; y /= z; }
                else x = y = -1;
            }
        };


        template <typename Mapper>
        __global__ void buildWarpMapsKernel(int tl_u, int tl_v, int cols, int rows,
                                            PtrStepf map_x, PtrStepf map_y)
        {
            int du = blockIdx.x * blockDim.x + threadIdx.x;
            int dv = blockIdx.y * blockDim.y + threadIdx.y;
            if (du < cols && dv < rows)
            {
                float u = tl_u + du;
                float v = tl_v + dv;
                float x, y;
                Mapper::mapBackward(u, v, x, y);
                map_x.ptr(dv)[du] = x;
                map_y.ptr(dv)[du] = y;
            }
        }


        void buildWarpPlaneMaps(int tl_u, int tl_v, PtrStepSzf map_x, PtrStepSzf map_y,
                                const float k_rinv[9], const float r_kinv[9], const float t[3],
                                float scale, hipStream_t stream)
        {
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::ck_rinv), k_rinv, 9*sizeof(float)));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cr_kinv), r_kinv, 9*sizeof(float)));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::ct), t, 3*sizeof(float)));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cscale), &scale, sizeof(float)));

            int cols = map_x.cols;
            int rows = map_x.rows;

            dim3 threads(32, 8);
            dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

            buildWarpMapsKernel<PlaneMapper><<<grid,threads>>>(tl_u, tl_v, cols, rows, map_x, map_y);
            cudaSafeCall(hipGetLastError());
            if (stream == 0)
                cudaSafeCall(hipDeviceSynchronize());
        }


        void buildWarpCylindricalMaps(int tl_u, int tl_v, PtrStepSzf map_x, PtrStepSzf map_y,
                                      const float k_rinv[9], const float r_kinv[9], float scale,
                                      hipStream_t stream)
        {
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::ck_rinv), k_rinv, 9*sizeof(float)));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cr_kinv), r_kinv, 9*sizeof(float)));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cscale), &scale, sizeof(float)));

            int cols = map_x.cols;
            int rows = map_x.rows;

            dim3 threads(32, 8);
            dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

            buildWarpMapsKernel<CylindricalMapper><<<grid,threads>>>(tl_u, tl_v, cols, rows, map_x, map_y);
            cudaSafeCall(hipGetLastError());
            if (stream == 0)
                cudaSafeCall(hipDeviceSynchronize());
        }


        void buildWarpSphericalMaps(int tl_u, int tl_v, PtrStepSzf map_x, PtrStepSzf map_y,
                                    const float k_rinv[9], const float r_kinv[9], float scale,
                                    hipStream_t stream)
        {
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::ck_rinv), k_rinv, 9*sizeof(float)));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cr_kinv), r_kinv, 9*sizeof(float)));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cscale), &scale, sizeof(float)));

            int cols = map_x.cols;
            int rows = map_x.rows;

            dim3 threads(32, 8);
            dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

            buildWarpMapsKernel<SphericalMapper><<<grid,threads>>>(tl_u, tl_v, cols, rows, map_x, map_y);
            cudaSafeCall(hipGetLastError());
            if (stream == 0)
                cudaSafeCall(hipDeviceSynchronize());
        }
    } // namespace imgproc
}}} // namespace cv { namespace gpu { namespace cudev {


#endif /* CUDA_DISABLER */
