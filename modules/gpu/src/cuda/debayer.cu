#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/gpu/device/common.hpp"
#include "opencv2/gpu/device/vec_traits.hpp"
#include "opencv2/gpu/device/vec_math.hpp"
#include "opencv2/gpu/device/limits.hpp"
#include "opencv2/gpu/device/color.hpp"

namespace cv { namespace gpu { namespace device
{
    template <typename T> struct Bayer2BGR;

    template <> struct Bayer2BGR<uchar>
    {
        uchar3 res0;
        uchar3 res1;
        uchar3 res2;
        uchar3 res3;

        __device__ void apply(const PtrStepSzb& src, int s_x, int s_y, bool blue_last, bool start_with_green)
        {
            uchar4 patch[3][3];
            patch[0][1] = ((const uchar4*) src.ptr(s_y - 1))[s_x];
            patch[0][0] = ((const uchar4*) src.ptr(s_y - 1))[::max(s_x - 1, 0)];
            patch[0][2] = ((const uchar4*) src.ptr(s_y - 1))[::min(s_x + 1, ((src.cols + 3) >> 2) - 1)];

            patch[1][1] = ((const uchar4*) src.ptr(s_y))[s_x];
            patch[1][0] = ((const uchar4*) src.ptr(s_y))[::max(s_x - 1, 0)];
            patch[1][2] = ((const uchar4*) src.ptr(s_y))[::min(s_x + 1, ((src.cols + 3) >> 2) - 1)];

            patch[2][1] = ((const uchar4*) src.ptr(s_y + 1))[s_x];
            patch[2][0] = ((const uchar4*) src.ptr(s_y + 1))[::max(s_x - 1, 0)];
            patch[2][2] = ((const uchar4*) src.ptr(s_y + 1))[::min(s_x + 1, ((src.cols + 3) >> 2) - 1)];

            if ((s_y & 1) ^ start_with_green)
            {
                const int t0 = (patch[0][1].x + patch[2][1].x + 1) >> 1;
                const int t1 = (patch[1][0].w + patch[1][1].y + 1) >> 1;

                const int t2 = (patch[0][1].x + patch[0][1].z + patch[2][1].x + patch[2][1].z + 2) >> 2;
                const int t3 = (patch[0][1].y + patch[1][1].x + patch[1][1].z + patch[2][1].y + 2) >> 2;

                const int t4 = (patch[0][1].z + patch[2][1].z + 1) >> 1;
                const int t5 = (patch[1][1].y + patch[1][1].w + 1) >> 1;

                const int t6 = (patch[0][1].z + patch[0][2].x + patch[2][1].z + patch[2][2].x + 2) >> 2;
                const int t7 = (patch[0][1].w + patch[1][1].z + patch[1][2].x + patch[2][1].w + 2) >> 2;

                if ((s_y & 1) ^ blue_last)
                {
                    res0.x = t1;
                    res0.y = patch[1][1].x;
                    res0.z = t0;

                    res1.x = patch[1][1].y;
                    res1.y = t3;
                    res1.z = t2;

                    res2.x = t5;
                    res2.y = patch[1][1].z;
                    res2.z = t4;

                    res3.x = patch[1][1].w;
                    res3.y = t7;
                    res3.z = t6;
                }
                else
                {
                    res0.x = t0;
                    res0.y = patch[1][1].x;
                    res0.z = t1;

                    res1.x = t2;
                    res1.y = t3;
                    res1.z = patch[1][1].y;

                    res2.x = t4;
                    res2.y = patch[1][1].z;
                    res2.z = t5;

                    res3.x = t6;
                    res3.y = t7;
                    res3.z = patch[1][1].w;
                }
            }
            else
            {
                const int t0 = (patch[0][0].w + patch[0][1].y + patch[2][0].w + patch[2][1].y + 2) >> 2;
                const int t1 = (patch[0][1].x + patch[1][0].w + patch[1][1].y + patch[2][1].x + 2) >> 2;

                const int t2 = (patch[0][1].y + patch[2][1].y + 1) >> 1;
                const int t3 = (patch[1][1].x + patch[1][1].z + 1) >> 1;

                const int t4 = (patch[0][1].y + patch[0][1].w + patch[2][1].y + patch[2][1].w + 2) >> 2;
                const int t5 = (patch[0][1].z + patch[1][1].y + patch[1][1].w + patch[2][1].z + 2) >> 2;

                const int t6 = (patch[0][1].w + patch[2][1].w + 1) >> 1;
                const int t7 = (patch[1][1].z + patch[1][2].x + 1) >> 1;

                if ((s_y & 1) ^ blue_last)
                {
                    res0.x = patch[1][1].x;
                    res0.y = t1;
                    res0.z = t0;

                    res1.x = t3;
                    res1.y = patch[1][1].y;
                    res1.z = t2;

                    res2.x = patch[1][1].z;
                    res2.y = t5;
                    res2.z = t4;

                    res3.x = t7;
                    res3.y = patch[1][1].w;
                    res3.z = t6;
                }
                else
                {
                    res0.x = t0;
                    res0.y = t1;
                    res0.z = patch[1][1].x;

                    res1.x = t2;
                    res1.y = patch[1][1].y;
                    res1.z = t3;

                    res2.x = t4;
                    res2.y = t5;
                    res2.z = patch[1][1].z;

                    res3.x = t6;
                    res3.y = patch[1][1].w;
                    res3.z = t7;
                }
            }
        }
    };

    template <typename D> __device__ __forceinline__ D toDst(const uchar3& pix);
    template <> __device__ __forceinline__ uchar toDst<uchar>(const uchar3& pix)
    {
        typename bgr_to_gray_traits<uchar>::functor_type f = bgr_to_gray_traits<uchar>::create_functor();
        return f(pix);
    }
    template <> __device__ __forceinline__ uchar3 toDst<uchar3>(const uchar3& pix)
    {
        return pix;
    }
    template <> __device__ __forceinline__ uchar4 toDst<uchar4>(const uchar3& pix)
    {
        return make_uchar4(pix.x, pix.y, pix.z, 255);
    }

    template <typename D>
    __global__ void Bayer2BGR_8u(const PtrStepSzb src, PtrStep<D> dst, const bool blue_last, const bool start_with_green)
    {
        const int s_x = blockIdx.x * blockDim.x + threadIdx.x;
        int s_y = blockIdx.y * blockDim.y + threadIdx.y;

        if (s_y >= src.rows || (s_x << 2) >= src.cols)
            return;

        s_y = ::min(::max(s_y, 1), src.rows - 2);

        Bayer2BGR<uchar> bayer;
        bayer.apply(src, s_x, s_y, blue_last, start_with_green);

        const int d_x = (blockIdx.x * blockDim.x + threadIdx.x) << 2;
        const int d_y = blockIdx.y * blockDim.y + threadIdx.y;

        dst(d_y, d_x) = toDst<D>(bayer.res0);
        if (d_x + 1 < src.cols)
            dst(d_y, d_x + 1) = toDst<D>(bayer.res1);
        if (d_x + 2 < src.cols)
            dst(d_y, d_x + 2) = toDst<D>(bayer.res2);
        if (d_x + 3 < src.cols)
            dst(d_y, d_x + 3) = toDst<D>(bayer.res3);
    }

    template <> struct Bayer2BGR<ushort>
    {
        ushort3 res0;
        ushort3 res1;

        __device__ void apply(const PtrStepSzb& src, int s_x, int s_y, bool blue_last, bool start_with_green)
        {
            ushort2 patch[3][3];
            patch[0][1] = ((const ushort2*) src.ptr(s_y - 1))[s_x];
            patch[0][0] = ((const ushort2*) src.ptr(s_y - 1))[::max(s_x - 1, 0)];
            patch[0][2] = ((const ushort2*) src.ptr(s_y - 1))[::min(s_x + 1, ((src.cols + 1) >> 1) - 1)];

            patch[1][1] = ((const ushort2*) src.ptr(s_y))[s_x];
            patch[1][0] = ((const ushort2*) src.ptr(s_y))[::max(s_x - 1, 0)];
            patch[1][2] = ((const ushort2*) src.ptr(s_y))[::min(s_x + 1, ((src.cols + 1) >> 1) - 1)];

            patch[2][1] = ((const ushort2*) src.ptr(s_y + 1))[s_x];
            patch[2][0] = ((const ushort2*) src.ptr(s_y + 1))[::max(s_x - 1, 0)];
            patch[2][2] = ((const ushort2*) src.ptr(s_y + 1))[::min(s_x + 1, ((src.cols + 1) >> 1) - 1)];

            if ((s_y & 1) ^ start_with_green)
            {
                const int t0 = (patch[0][1].x + patch[2][1].x + 1) >> 1;
                const int t1 = (patch[1][0].y + patch[1][1].y + 1) >> 1;

                const int t2 = (patch[0][1].x + patch[0][2].x + patch[2][1].x + patch[2][2].x + 2) >> 2;
                const int t3 = (patch[0][1].y + patch[1][1].x + patch[1][2].x + patch[2][1].y + 2) >> 2;

                if ((s_y & 1) ^ blue_last)
                {
                    res0.x = t1;
                    res0.y = patch[1][1].x;
                    res0.z = t0;

                    res1.x = patch[1][1].y;
                    res1.y = t3;
                    res1.z = t2;
                }
                else
                {
                    res0.x = t0;
                    res0.y = patch[1][1].x;
                    res0.z = t1;

                    res1.x = t2;
                    res1.y = t3;
                    res1.z = patch[1][1].y;
                }
            }
            else
            {
                const int t0 = (patch[0][0].y + patch[0][1].y + patch[2][0].y + patch[2][1].y + 2) >> 2;
                const int t1 = (patch[0][1].x + patch[1][0].y + patch[1][1].y + patch[2][1].x + 2) >> 2;

                const int t2 = (patch[0][1].y + patch[2][1].y + 1) >> 1;
                const int t3 = (patch[1][1].x + patch[1][2].x + 1) >> 1;

                if ((s_y & 1) ^ blue_last)
                {
                    res0.x = patch[1][1].x;
                    res0.y = t1;
                    res0.z = t0;

                    res1.x = t3;
                    res1.y = patch[1][1].y;
                    res1.z = t2;
                }
                else
                {
                    res0.x = t0;
                    res0.y = t1;
                    res0.z = patch[1][1].x;

                    res1.x = t2;
                    res1.y = patch[1][1].y;
                    res1.z = t3;
                }
            }
        }
    };

    template <typename D> __device__ __forceinline__ D toDst(const ushort3& pix);
    template <> __device__ __forceinline__ ushort toDst<ushort>(const ushort3& pix)
    {
        typename bgr_to_gray_traits<ushort>::functor_type f = bgr_to_gray_traits<ushort>::create_functor();
        return f(pix);
    }
    template <> __device__ __forceinline__ ushort3 toDst<ushort3>(const ushort3& pix)
    {
        return pix;
    }
    template <> __device__ __forceinline__ ushort4 toDst<ushort4>(const ushort3& pix)
    {
        return make_ushort4(pix.x, pix.y, pix.z, numeric_limits<ushort>::max());
    }

    template <typename D>
    __global__ void Bayer2BGR_16u(const PtrStepSzb src, PtrStep<D> dst, const bool blue_last, const bool start_with_green)
    {
        const int s_x = blockIdx.x * blockDim.x + threadIdx.x;
        int s_y = blockIdx.y * blockDim.y + threadIdx.y;

        if (s_y >= src.rows || (s_x << 1) >= src.cols)
            return;

        s_y = ::min(::max(s_y, 1), src.rows - 2);

        Bayer2BGR<ushort> bayer;
        bayer.apply(src, s_x, s_y, blue_last, start_with_green);

        const int d_x = (blockIdx.x * blockDim.x + threadIdx.x) << 1;
        const int d_y = blockIdx.y * blockDim.y + threadIdx.y;

        dst(d_y, d_x) = toDst<D>(bayer.res0);
        if (d_x + 1 < src.cols)
            dst(d_y, d_x + 1) = toDst<D>(bayer.res1);
    }

    template <int cn>
    void Bayer2BGR_8u_gpu(PtrStepSzb src, PtrStepSzb dst, bool blue_last, bool start_with_green, hipStream_t stream)
    {
        typedef typename TypeVec<uchar, cn>::vec_type dst_t;

        const dim3 block(32, 8);
        const dim3 grid(divUp(src.cols, 4 * block.x), divUp(src.rows, block.y));

        cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(Bayer2BGR_8u<dst_t>), hipFuncCachePreferL1) );

        Bayer2BGR_8u<dst_t><<<grid, block, 0, stream>>>(src, (PtrStepSz<dst_t>)dst, blue_last, start_with_green);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    template <int cn>
    void Bayer2BGR_16u_gpu(PtrStepSzb src, PtrStepSzb dst, bool blue_last, bool start_with_green, hipStream_t stream)
    {
        typedef typename TypeVec<ushort, cn>::vec_type dst_t;

        const dim3 block(32, 8);
        const dim3 grid(divUp(src.cols, 2 * block.x), divUp(src.rows, block.y));

        cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(Bayer2BGR_16u<dst_t>), hipFuncCachePreferL1) );

        Bayer2BGR_16u<dst_t><<<grid, block, 0, stream>>>(src, (PtrStepSz<dst_t>)dst, blue_last, start_with_green);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    template void Bayer2BGR_8u_gpu<1>(PtrStepSzb src, PtrStepSzb dst, bool blue_last, bool start_with_green, hipStream_t stream);
    template void Bayer2BGR_8u_gpu<3>(PtrStepSzb src, PtrStepSzb dst, bool blue_last, bool start_with_green, hipStream_t stream);
    template void Bayer2BGR_8u_gpu<4>(PtrStepSzb src, PtrStepSzb dst, bool blue_last, bool start_with_green, hipStream_t stream);

    template void Bayer2BGR_16u_gpu<1>(PtrStepSzb src, PtrStepSzb dst, bool blue_last, bool start_with_green, hipStream_t stream);
    template void Bayer2BGR_16u_gpu<3>(PtrStepSzb src, PtrStepSzb dst, bool blue_last, bool start_with_green, hipStream_t stream);
    template void Bayer2BGR_16u_gpu<4>(PtrStepSzb src, PtrStepSzb dst, bool blue_last, bool start_with_green, hipStream_t stream);
}}}

#endif /* CUDA_DISABLER */
